#include "hip/hip_runtime.h"
/**
 * @author Marco Maggioni 
 * @author Alan Perez-Rathke 
 *
 * @date April 7, 2011 
 *
 * Department of Computer Science
 * University of Illinois at Chicago 
 */

#include "PkBuild.h"
#include "PkTypes.h"
#include "PkAssert.h"
#include "PkMiscUtil.h"
#include "PkGlobalData.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cutil.h>

// Switch which controls whether to use GPU to convert to CPU memory formats
#define Pk_ENABLE_CUDA_MAP_GPU_TO_CPU_INTERSECTION 1

// The number of CUDA threads in block x and y dimensions
#define Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X 16
#define Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y 16
#define Pk_CUDA_NUM_TOTAL_INTERSECTION_THREADS_IN_BLOCK 256
#if (Pk_CUDA_NUM_TOTAL_INTERSECTION_THREADS_IN_BLOCK != (Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y))
	#error Number of total intersection threads must be the product of number of threads in x and y
#endif

// CUDA assert
#if Pk_ENABLE_ASSERT
	#define PkCUDAAssert( X ) \
		if ( !(X) ) { printf( "CUDA Thread %d:%d failed assert at %s:%d!", blockIdx.x, threadIdx.x, __FILE__, __LINE__ ); return; }
#else
	#define PkCUDAAssert( X ) // do nothing
#endif

// The underlying block primitive for representing a bitset
typedef PkPooledRawBitSetArray::block_type PkCUDABlockType;
// A basic byte type
typedef PkPooledRawBitSetArray::byte_type  PkCUDAByteType;
// A size type for memory allocations, etc.
typedef PkPooledRawBitSetArray::size_type  PkCUDASizeType;
// A size for counting bits
typedef unsigned short                     PkCUDABitCountType;

// The number of bits that a CUDA block type can store
#define Pk_CUDA_BITS_PER_BLOCK PkPooledRawBitSetArray::bits_per_block

namespace Pk
{

///**********************************************************

// @return - CPU pointer to data from GPU, client is responsbile for freeing allocated memory
template < typename tData >
tData* CUDACopyFromGPUToCPU
(
  const tData* gpu_data
, const size_t num_data_values
)
{
	// Copy bit counts back to CPU
	tData * cpu_data = (tData*) malloc( sizeof( tData ) * num_data_values );
	PkAssert( NULL != cpu_data );
	hipMemcpy(
	  cpu_data
	, gpu_data
	, sizeof( tData ) * num_data_values
	, hipMemcpyDeviceToHost
	);

	return cpu_data;
}

// @return - CPU pointer to data from GPU, release CPU memory, client is responsbile for freeing CPU allocated memory
template < typename tData >
tData* CUDACopyFromGPUToCPUAndReleaseGPU
(
  const tData*& gpu_data
, const size_t num_data_values
)
{
	tData* cpu_data = CUDACopyFromGPUToCPU<tData>( gpu_data, num_data_values );
	PkAssert( NULL != cpu_data );
	hipFree( (void*)gpu_data );
	gpu_data = (const tData*)NULL;
	return cpu_data;
}

///**********************************************************

// GPU kernel for calculating the intersection
__global__ void CUDASwizzledSiblingIntersectionKernel(
  PkCUDABlockType * gpu_sib_sets_matrix_a
, PkCUDABlockType * gpu_sib_sets_matrix_b
, PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
)
{
	// Get position in the overall computation matrix
	const unsigned int locus_a_offset = blockIdx.y * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y + threadIdx.y;
	const unsigned int locus_b_offset = blockIdx.x * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X + threadIdx.x;

	if ( ( locus_a_offset < num_sib_sets_at_locus_a ) && ( locus_b_offset < num_sib_sets_at_locus_b ) )
	{
		// Compute the intersection
		for ( int itr_block=0; itr_block<num_bit_blocks_per_sib_set; ++itr_block )
		{
			gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * itr_block) + (num_sib_sets_at_locus_b * locus_a_offset) + locus_b_offset ] = 
				  gpu_sib_sets_matrix_a[ ( num_sib_sets_at_locus_a * itr_block ) + locus_a_offset ] 
				& gpu_sib_sets_matrix_b[ ( num_sib_sets_at_locus_b * itr_block ) + locus_b_offset ];
		}
	}
}

///**********************************************************

// GPU kernel for counting the number of bits in each intersection bit set
__global__ void CUDASwizzledCountBitsKernel(
  PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
)
{
	// Store byte to bit count mapping
	// @todo - move to shared memory!
	const PkCUDAByteType num_bits_in_byte_table[] = Pk_INIT_NUM_BITS_IN_BYTE_TABLE;

	// Get position in the overall computation matrix
	const unsigned int locus_a_offset = blockIdx.y * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y + threadIdx.y;
	const unsigned int locus_b_offset = blockIdx.x * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X + threadIdx.x;

	// Make sure we are in bounds
	if ( ( locus_a_offset < num_sib_sets_at_locus_a ) && ( locus_b_offset < num_sib_sets_at_locus_b ) )
	{
		// Initialize bit count to 0
		const size_t idx_bit_set = (num_sib_sets_at_locus_b * locus_a_offset) + locus_b_offset;
		gpu_sib_sets_matrix_intersection_bit_counts[ idx_bit_set ] = 0;

		// Count number of bits
		for ( int itr_block=0; itr_block<num_bit_blocks_per_sib_set; ++itr_block )
		{
			// Cast block to a byte buffer
			const PkCUDAByteType* byte_buffer = 
				(PkCUDAByteType*)(&(gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * itr_block) + idx_bit_set ]));
			
			// Iterate over bytes and count bits
			for ( int itr_byte=0; itr_byte<sizeof(PkCUDABlockType); ++itr_byte )
			{
				gpu_sib_sets_matrix_intersection_bit_counts[ idx_bit_set ] += num_bits_in_byte_table[ byte_buffer[ itr_byte ] ];
			}
		}
	}
}

// @return - Pointer to GPU array containing bit counts, client is responsible for freeing and for transferring to CPU
PkCUDABitCountType * CUDACountIntersectionBits(
  const PkCUDABlockType* gpu_sib_sets_matrix_intersection
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
, const dim3 dimComputingGrid
, const dim3 dimComputingBlock
)
{
	// Determine number of values necessary for storing bit counts and hash values
	const size_t num_data_values = num_sib_sets_at_locus_a * num_sib_sets_at_locus_b;
	
	// Allocate bit counts array
	PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts = NULL;
	hipMalloc((void**)&gpu_sib_sets_matrix_intersection_bit_counts, num_data_values*sizeof(PkCUDABitCountType) );		
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_bit_counts );

	// Wait for intersections to finish before executing bit counts kernel
	PkVerify( hipSuccess == hipDeviceSynchronize() );	

	// Count the number of bits in each sibling set
	CUDASwizzledCountBitsKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_matrix_intersection_bit_counts
		, gpu_sib_sets_matrix_intersection
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		);

	return gpu_sib_sets_matrix_intersection_bit_counts;
}

///**********************************************************

// From boost::hash

template <class T>
__device__ std::size_t CUDAHashValueUnsigned(T val)
{
     const int size_t_bits = std::numeric_limits<std::size_t>::digits;
     // ceiling(std::numeric_limits<T>::digits / size_t_bits) - 1
     const int length = (std::numeric_limits<T>::digits - 1)
         / size_t_bits;

     std::size_t seed = 0;

     // Hopefully, this loop can be unrolled.
     for(unsigned int i = length * size_t_bits; i > 0; i -= size_t_bits)
     {
         seed ^= (std::size_t) (val >> i) + (seed<<6) + (seed>>2);
     }
     seed ^= (std::size_t) val + (seed<<6) + (seed>>2);

     return seed;
}

template <class T>
__device__ void CUDAHashCombineUnsigned(std::size_t& seed, T const& v)
{
    seed ^= CUDAHashValueUnsigned(v) + 0x9e3779b9 + (seed<<6) + (seed>>2);
}

template <class It>
__device__ void CUDAHashRangeUnsigned(std::size_t& seed, It first, It last)
{
	for(; first != last; ++first)
    {
		CUDAHashCombineUnsigned(seed, *first);
	}
}

// GPU kernel for mapping each intersection bit set to a hash value
__global__ void CUDASwizzledHashBlocksKernel(
  std::size_t * gpu_sib_sets_matrix_intersection_hashes
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
)
{
	// Get position in the overall computation matrix
	const unsigned int locus_a_offset = blockIdx.y * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y + threadIdx.y;
	const unsigned int locus_b_offset = blockIdx.x * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X + threadIdx.x;

	// Make sure we are in bounds
	if ( ( locus_a_offset < num_sib_sets_at_locus_a ) && ( locus_b_offset < num_sib_sets_at_locus_b ) )
	{
		// Initialize bit count to 0
		const size_t idx_bit_set = (num_sib_sets_at_locus_b * locus_a_offset) + locus_b_offset;

		// Hashing routine below is based on Boost's hash library

		// Initialize our hashed bit set seed to 0
		std::size_t hashed_bit_set = 0;

		// Hash combine each block
		for ( int itr_block=0; itr_block<num_bit_blocks_per_sib_set; ++itr_block )
		{
			// Get block value to hash
			const PkCUDABlockType block =
				gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * itr_block) + idx_bit_set ];
			
			CUDAHashCombineUnsigned( hashed_bit_set, block );
		}

		// Hash combine with the bit count
		CUDAHashCombineUnsigned( hashed_bit_set, gpu_sib_sets_matrix_intersection_bit_counts[ idx_bit_set ] );

		// Store resulting bit set hash
		gpu_sib_sets_matrix_intersection_hashes[ idx_bit_set ] = hashed_bit_set;
	}
}

// @return - Pointer to GPU array containing bit set hashes, client is responsible for freeing and for transferring to CPU
std::size_t * CUDAComputeIntersectionHashes(
  const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
, const dim3 dimComputingGrid
, const dim3 dimComputingBlock
)
{
	// Determine number of values necessary for storing bit counts and hash values
	const size_t num_data_values = num_sib_sets_at_locus_a * num_sib_sets_at_locus_b;
	
	// Allocate bit counts array
	std::size_t * gpu_sib_sets_matrix_intersection_hashes = NULL;
	hipMalloc((void**)&gpu_sib_sets_matrix_intersection_hashes, num_data_values*sizeof(std::size_t) );		
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_hashes );

	// Count the number of bits in each sibling set
	CUDASwizzledHashBlocksKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_matrix_intersection_hashes
		, gpu_sib_sets_matrix_intersection
		, gpu_sib_sets_matrix_intersection_bit_counts
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		);

	return gpu_sib_sets_matrix_intersection_hashes;
}

///**********************************************************

// GPU kernel for transposing swizzled bits
// Note: Assumes hipMemset was used to initialize the transpose matrix to all 0's
__global__ void CUDASwizzledTransposeKernel(
  PkCUDABlockType * gpu_sib_sets_matrix_intersection_transpose
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
, const size_t num_bits_per_sib_set
)
{
	// Get position in the overall computation matrix
	const size_t locus_a_offset = blockIdx.y * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y + threadIdx.y;
	const size_t locus_b_offset = blockIdx.x * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X + threadIdx.x;

	// Make sure we are in bounds
	if ( ( locus_a_offset < num_sib_sets_at_locus_a ) && ( locus_b_offset < num_sib_sets_at_locus_b ) )
	{
		// Store bit set offset
		const size_t idx_bit_set = (num_sib_sets_at_locus_b * locus_a_offset) + locus_b_offset;

		// Early out if bit set is empty
		if ( 0 == gpu_sib_sets_matrix_intersection_bit_counts[ idx_bit_set ] )
		{
			return;
		}

		// Transpose each bit set
		for ( size_t itr_block=0; itr_block<num_bit_blocks_per_sib_set; ++itr_block )
		{
			// Transpose block
			const PkCUDABlockType block = 
				gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * itr_block) + idx_bit_set ];
			
			// Early out if block is empty
			// @todo: branching may not be warranted
			if ( 0 == block )
			{
				continue;
			}

			// itr_bit is relative to this bit offset:
			const size_t bit_offset = Pk_CUDA_BITS_PER_BLOCK * itr_block;

			// Transpose each bit
			for ( size_t itr_bit=0; (itr_bit < Pk_CUDA_BITS_PER_BLOCK) && ((bit_offset+itr_bit)<num_bits_per_sib_set); ++itr_bit )
			{
				if ( block & (1 << itr_bit) )
				{
					// Note: locus_a_offset is the "row", locus_b_offset is the "column", and itr_block is the "depth"
					// To transpose, we want each vertical plane to become a horizontal plane.
					// This can be accomplished by simply swapping row and depth
					// Note: (locus_a_offset/Pk_CUDA_BITS_PER_BLOCK) is the new block offset (i.e. - "itr_block")
					// -and- (locus_a_offset%Pk_CUDA_BITS_PER_BLOCK) is the bit offset within that block
					// -and- (bit_offset+itr_bit) is the new row (i.e. - what used to be "locus_a_offset")
					const size_t idx_trans_block = (num_bits_per_sib_set*num_sib_sets_at_locus_b * (locus_a_offset/Pk_CUDA_BITS_PER_BLOCK)) + (num_sib_sets_at_locus_b * (bit_offset+itr_bit)) + locus_b_offset;
					gpu_sib_sets_matrix_intersection_transpose[ idx_trans_block ] |= ( 1 << (locus_a_offset%Pk_CUDA_BITS_PER_BLOCK) );
				}
			} // end iteration over bits
		} // end iteration over blocks
	} // end bounds check
}

// @return - Pointer to GPU matrix containing transpose of intersections, client is responsible for freeing
PkCUDABlockType* CUDATransposeIntersections(
  const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
, const size_t num_bits_per_sib_set
, const size_t num_bit_blocks_for_locus_a
, const dim3 dimComputingGrid
, const dim3 dimComputingBlock
)
{
	// Allocate a matrix to store the transpose
	const size_t num_matrix_transpose_intersection_bytes = num_bits_per_sib_set * num_sib_sets_at_locus_b * num_bit_blocks_for_locus_a * sizeof( PkCUDABlockType );
	PkCUDABlockType * gpu_sib_sets_matrix_intersection_transpose = NULL;
	hipMalloc((void**)&gpu_sib_sets_matrix_intersection_transpose, num_matrix_transpose_intersection_bytes );		
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_transpose );

	// Zero out the transpose with hipMemset
	PkVerify( hipSuccess == hipMemset( (void*)gpu_sib_sets_matrix_intersection_transpose, 0, num_matrix_transpose_intersection_bytes ) );
	PkVerify( hipSuccess == hipDeviceSynchronize() );	

	// Note: Assumes hipMemset was used to initialize the transpose matrix to all 0's
	CUDASwizzledTransposeKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_matrix_intersection_transpose
		, gpu_sib_sets_matrix_intersection
		, gpu_sib_sets_matrix_intersection_bit_counts
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		, num_bits_per_sib_set
	);

	return gpu_sib_sets_matrix_intersection_transpose;
}

///**********************************************************

// GPU kernel for transposing swizzled bits
__global__ void CUDASwizzledCountSubsetsKernel(
  unsigned int * gpu_sib_sets_matrix_intersection_subsets_counts 
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection_transpose
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
, const size_t num_bits_per_sib_set
, const size_t num_bit_blocks_for_locus_a
)
{
	// Store byte to bit count mapping
	// @todo - move to shared memory!
	const PkCUDAByteType num_bits_in_byte_table[] = Pk_INIT_NUM_BITS_IN_BYTE_TABLE;

	// Get position in the overall computation matrix
	const size_t locus_a_offset = blockIdx.y * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y + threadIdx.y;
	const size_t locus_b_offset = blockIdx.x * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X + threadIdx.x;

	if ( ( locus_a_offset < num_sib_sets_at_locus_a ) && ( locus_b_offset < num_sib_sets_at_locus_b ) )
	{
		// Determine which sibling set we're working with
		const size_t idx_bit_set = (num_sib_sets_at_locus_b * locus_a_offset) + locus_b_offset;

		// Initialize subset count to 0
		gpu_sib_sets_matrix_intersection_subsets_counts[ idx_bit_set ] = 0;

		// Initialize first bit offsets
		size_t first_block = num_bit_blocks_per_sib_set;
		size_t first_bit_in_first_block = Pk_CUDA_BITS_PER_BLOCK;
		size_t first_bit_in_transpose = Pk_CUDA_BITS_PER_BLOCK * num_bit_blocks_per_sib_set;

		// Determine first horizontal plane to start intersecting
		for ( first_block=0; first_block<num_bit_blocks_per_sib_set; ++first_block )
		{
			const PkCUDABlockType block =
				gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * first_block) + idx_bit_set ];
			
			// Skip blocks that are empty
			// @todo: branching may not be warranted
			if ( 0 == block )
			{
				continue;
			}

			// the bit iterator below is relative to this offset
			const size_t bit_offset = first_block * Pk_CUDA_BITS_PER_BLOCK;

			// test to see if this block contains the first true bit
			for ( 
				  first_bit_in_first_block=0
				; (first_bit_in_first_block<Pk_CUDA_BITS_PER_BLOCK) && (( bit_offset + first_bit_in_first_block ) < num_bits_per_sib_set)
				; ++first_bit_in_first_block
				)
			{
				if ( block & (1 << first_bit_in_first_block) )
				{
					first_bit_in_transpose = bit_offset + first_bit_in_first_block;
					break;
				}
			} // end iteration over bits

			// End iteration over blocks once we found the first bit
			if ( first_bit_in_first_block != Pk_CUDA_BITS_PER_BLOCK )
			{
				break;
			}
		} // end iteration over blocks to find first bit

		// Early out if first bit is invalid (i.e. - all bits are 0)
		// @todo: could also just use bit counts then would never get here
		if ( first_bit_in_transpose == num_bits_per_sib_set )
		{
			return;
		}

		// Iterate over every column
		for ( size_t itr_trans_locus_b=0; itr_trans_locus_b<num_sib_sets_at_locus_b; ++itr_trans_locus_b )
		{
			// Iterate over every block in this horizontal slize of the transpose

			for ( size_t itr_trans_locus_a_block=0; itr_trans_locus_a_block<num_bit_blocks_for_locus_a; ++itr_trans_locus_a_block )
			{
				// Initialize intersection result to first block
				PkCUDABlockType transposeBlockIntersectionResult =
					gpu_sib_sets_matrix_intersection_transpose[ (num_bits_per_sib_set*num_sib_sets_at_locus_b * itr_trans_locus_a_block) + (num_sib_sets_at_locus_b*first_bit_in_transpose) + itr_trans_locus_b ];

				// Intersect with remaining bits
				size_t next_bit_in_next_block = first_bit_in_first_block+1;
				for ( size_t next_block=first_block; next_block<num_bit_blocks_per_sib_set; ++next_block )
				{
					const PkCUDABlockType block =
						gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * next_block) + idx_bit_set ];
			
					// Skip zero blocks
					// @todo: branching may not be warranted
					if ( 0 == block )
					{
						next_bit_in_next_block = 0;
						continue;
					}

					// Find next true bit
					for ( ; next_bit_in_next_block<Pk_CUDA_BITS_PER_BLOCK; ++next_bit_in_next_block )
					{
						if ( block & (1 << next_bit_in_next_block) )
						{
							// determine row in transpose
							const size_t next_bit_in_transpose = (Pk_CUDA_BITS_PER_BLOCK*next_block) + next_bit_in_next_block;

							// update transpose intersection result
							transposeBlockIntersectionResult &=
								gpu_sib_sets_matrix_intersection_transpose[ (num_bits_per_sib_set*num_sib_sets_at_locus_b * itr_trans_locus_a_block) + (num_sib_sets_at_locus_b*next_bit_in_transpose) + itr_trans_locus_b ];
						}
					} // end iteration over bits (non-transpose)

					// Reset bit iterator for next block
					next_bit_in_next_block = 0;
				} // end iteration over the remaining vertical column (non-transpose)

				// Count the number of bits in this column
				// Cast block to a byte buffer
				const PkCUDAByteType* byte_buffer = (PkCUDAByteType*)(&transposeBlockIntersectionResult);
			
				// Iterate over bytes and count bits
				for ( size_t itr_byte=0; itr_byte<sizeof(PkCUDABlockType); ++itr_byte )
				{
					gpu_sib_sets_matrix_intersection_subsets_counts[ idx_bit_set ] += num_bits_in_byte_table[ byte_buffer[ itr_byte ] ];
				}
			} // end transpose iteration over locus a blocks (depth)
		} // end transpose iteration over locus b (the columns)

	} // end bounds check
}

// @return - Pointer to GPU subsets counts, client is responsible for freeing memory
unsigned int * CUDASwizzledCountSubsets(
  const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection_transpose
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_bit_blocks_per_sib_set
, const size_t num_bits_per_sib_set
, const size_t num_bit_blocks_for_locus_a
, const dim3 dimComputingGrid
, const dim3 dimComputingBlock
)
{
	// Determine number of values necessary for storing bit counts and hash values
	const size_t num_data_values = num_sib_sets_at_locus_a * num_sib_sets_at_locus_b;
	
	// Allocate bit counts array
	unsigned int * gpu_sib_sets_matrix_intersection_subsets_counts = NULL;
	hipMalloc((void**)&gpu_sib_sets_matrix_intersection_subsets_counts, num_data_values*sizeof(unsigned int) );		
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_subsets_counts );

	// Make sure transpose is ready
	PkVerify( hipSuccess == hipDeviceSynchronize() );	

	CUDASwizzledCountSubsetsKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_matrix_intersection_subsets_counts 
		, gpu_sib_sets_matrix_intersection
		, gpu_sib_sets_matrix_intersection_transpose
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		, num_bits_per_sib_set
		, num_bit_blocks_for_locus_a
		);

	return gpu_sib_sets_matrix_intersection_subsets_counts;
}

///**********************************************************

// GPU kernel for swizzling the bit set blocks into a transpose format for better memory coalescing
__global__ void CUDASwizzleMatrixBytesKernel(
  PkCUDABlockType * gpu_sib_sets_swizzled_matrix
, const PkCUDABlockType * gpu_sib_sets_matrix
, const size_t num_sib_sets
, const size_t num_bit_blocks_per_sib_set
)
{
	// Make sure we only have a linear block structure
	// PkCUDAAssert( 0 == blockIdx.y )
	// PkCUDAAssert( 0 == threadIdx.y )

	// Linearize this thread to a number from [0, Pk_CUDA_NUM_TOTAL_INTERSECTION_THREADS-1]
	const size_t bit_block_offset = Pk_CUDA_NUM_TOTAL_INTERSECTION_THREADS_IN_BLOCK * blockIdx.x + threadIdx.x;
		 
	// Do nothing if we're out of bounds
	if ( bit_block_offset >= num_sib_sets * num_bit_blocks_per_sib_set )
	{
		// Make sure we only allocated enough blocks to cover the buffer size
		// PkCUDAAssert( 0 < threadIdx.x )
		return;
	}

	// Compute transposed coordinates

	// For original coordinates:
	// row_orig = bit_block_offset / num_bit_blocks_per_sib_set
	// col_orig = bit_block_offset % num_bit_blocks_per_sib_set
	// width_orig = num_bit_blocks_per_sib_set
	// orig_block_offset = width_orig * row_orig + col_orig
	//
	// For transposed coordinates:
	// row_trans = col_orig
	// col_trans = row_orig
	// width_trans = num_sib_sets
	// trans_block_offset = width_trans * row_trans + col_trans = num_sib_sets * col_orig + row_orig
	// trans_block_offset = num_sib_sets * (bit_block_offset % num_bit_blocks_per_sib_set) + ( bit_block_offset / num_bit_blocks_per_sib_set )
	gpu_sib_sets_swizzled_matrix[ ( num_sib_sets * ( bit_block_offset % num_bit_blocks_per_sib_set ) ) +
		( bit_block_offset / num_bit_blocks_per_sib_set ) ] = gpu_sib_sets_matrix[ bit_block_offset ];
}

PkCUDABlockType* CUDASwizzleSiblingSets(
  const PkCUDABlockType* gpu_sib_sets_matrix
, const size_t num_sib_sets
, const size_t num_bit_blocks_per_sib_set
)
{
	const size_t num_total_bit_blocks = num_sib_sets * num_bit_blocks_per_sib_set;
	const unsigned int num_threads_in_block_x = Pk_CUDA_NUM_TOTAL_INTERSECTION_THREADS_IN_BLOCK;
	const unsigned int num_threads_in_block_y = 1;
	const unsigned int num_blocks_x = num_total_bit_blocks / num_threads_in_block_x + ( ( num_total_bit_blocks % num_threads_in_block_x ) > 0 );
	const unsigned int num_blocks_y = 1;
	const dim3 dimComputingGrid( num_blocks_x, num_blocks_y );
	const dim3 dimComputingBlock( num_threads_in_block_x, num_threads_in_block_y );

	// Allocate space for swizzled buffer on GPU
	PkCUDABlockType* gpu_sib_sets_swizzled_matrix = NULL;
	hipMalloc( (void**)&gpu_sib_sets_swizzled_matrix, num_total_bit_blocks * sizeof( PkCUDABlockType ) );
	PkAssert( NULL != gpu_sib_sets_swizzled_matrix );

	// Execute GPU kernel
	CUDASwizzleMatrixBytesKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_swizzled_matrix
		, gpu_sib_sets_matrix
		, num_sib_sets
		, num_bit_blocks_per_sib_set
	);
	
	return gpu_sib_sets_swizzled_matrix;
}

///**********************************************************

// GPU kernel for converting GPU intersection format to CPU friendly format
__global__ void CUDAUnswizzleIntersectionMatrixKernel(
  PkCUDABlockType * gpu_sib_sets_matrix_intersection_for_cpu
, const PkCUDABlockType * gpu_sib_sets_matrix_intersection
, const size_t num_sib_sets_at_locus_a
, const size_t num_sib_sets_at_locus_b
, const size_t num_buffer_elements
)
{
	// Get position in the overall computation matrix
	unsigned int locus_a_offset = blockIdx.y * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y + threadIdx.y;
	unsigned int locus_b_offset = blockIdx.x * Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X + threadIdx.x;

	if ( ( locus_a_offset < num_sib_sets_at_locus_a ) && locus_b_offset < ( num_sib_sets_at_locus_b ) )
	{
		// Map to cpu formatted memory
		for ( int itr_buffer_element=0; itr_buffer_element<num_buffer_elements; ++itr_buffer_element )
		{
			gpu_sib_sets_matrix_intersection_for_cpu[ (num_sib_sets_at_locus_b*num_buffer_elements*locus_a_offset) + (num_buffer_elements*locus_b_offset) + itr_buffer_element ] =
				gpu_sib_sets_matrix_intersection[ (num_sib_sets_at_locus_a*num_sib_sets_at_locus_b * itr_buffer_element) + (num_sib_sets_at_locus_b * locus_a_offset) + locus_b_offset ];
		}
	}
}

void CUDAUnswizzleIntersectionMatrix(
	  PkPooledRawBitSetArray& outputSets
	, const PkPooledRawBitSetArray& partitionSets
	, const size_t num_matrix_intersection_bytes
	, const PkCUDABlockType* gpu_sib_sets_matrix_intersection
	, const dim3 dimComputingGrid
	, const dim3 dimComputingBlock
	)
{
	PkLogf( "\tBegin CUDA conversion from GPU memory format to CPU memory format.\n" );
	PkAssert( ( outputSets.size() * partitionSets.size() * outputSets.num_blocks() * sizeof( PkCUDABlockType ) ) 
		== num_matrix_intersection_bytes );
	PkAssert( outputSets.num_blocks() == partitionSets.num_blocks() );

	// Begin timer
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);				

	// Allocate another GPU intersection matrix to convert to CPU format
	PkCUDABlockType* gpu_sib_sets_matrix_intersection_for_cpu = NULL;
	hipMalloc((void**)&gpu_sib_sets_matrix_intersection_for_cpu, num_matrix_intersection_bytes );		
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_for_cpu );

	// Run GPU kernel to convert to CPU memory format
	CUDAUnswizzleIntersectionMatrixKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_matrix_intersection_for_cpu
		, gpu_sib_sets_matrix_intersection
		, outputSets.size()
		, partitionSets.size()
		, outputSets.num_blocks()
		);

	// Allocate CPU memory for intersection
	PkCUDABlockType* cpu_sib_sets_matrix_intersection = (PkCUDABlockType*) malloc( num_matrix_intersection_bytes );
	PkAssert( NULL != cpu_sib_sets_matrix_intersection );
	hipMemcpy(
		  cpu_sib_sets_matrix_intersection
		, gpu_sib_sets_matrix_intersection_for_cpu
		, num_matrix_intersection_bytes
		, hipMemcpyDeviceToHost
		);

	PkLogf( "\t\tCUDA conversion finished (%d ms) - now converting to PkBitSetArray.\n", (unsigned int)cutGetTimerValue(timer) );
	
	// Free temporary GPU resources
	hipFree( gpu_sib_sets_matrix_intersection_for_cpu );

	// Pass ownership to output pooled array
	outputSets.reinit(
		  cpu_sib_sets_matrix_intersection
		, true /* b_owned */
		, outputSets.num_bits()
		, outputSets.size() * partitionSets.size()
		, 2 /* num_default_reserved_pool_elements */
		);

	// Assert that total number of bytes is consistent
	PkAssert( outputSets.num_total_bytes() == num_matrix_intersection_bytes );

	// Stop timer
	cutStopTimer(timer);
	PkLogf( "\tEnd CUDA conversion from GPU memory format to CPU memory format (%d ms).\n", (unsigned int)cutGetTimerValue(timer) );
	cutDeleteTimer(timer);
}

///**********************************************************

// Push an intersection buffer to GPU
PkCUDABlockType* CUDAPushSiblingSetsToGPU( const PkPooledRawBitSetArray& sibling_sets )
{
	PkLogf( "\tPushing sibling set ID %x to GPU.\n", &sibling_sets );
	PkAssert( sizeof( PkPooledRawBitSetArray::block_type ) == sizeof( PkCUDABlockType ) );

	// Allocate GPU sibling sets matrix
	PkCUDABlockType* gpu_sib_sets_matrix = NULL;
	hipMalloc( (void**)&gpu_sib_sets_matrix, sibling_sets.num_total_bytes() );
	PkAssert( NULL != gpu_sib_sets_matrix );

	// Copy CPU sibling set chunks to GPU
	PkCUDASizeType byte_offset = 0;
	for ( int itr_chunk=0; itr_chunk < sibling_sets.get_chunks().size(); ++itr_chunk )
	{
		// Copy sibling sets to memory
		hipMemcpy(
			  &(((PkCUDAByteType*)gpu_sib_sets_matrix)[ byte_offset ]) /* dst_ptr */
			, sibling_sets.get_chunks()[ itr_chunk ].first /* src_ptr */
			, sibling_sets.get_chunks()[ itr_chunk ].second /* num_bytes */
			, hipMemcpyHostToDevice
			);
			byte_offset += sibling_sets.get_chunks()[ itr_chunk ].second;
	}
	// Assert we pushed every last byte!
	PkAssert( sibling_sets.num_total_bytes() == byte_offset );

	// Output GPU pointer
	return gpu_sib_sets_matrix;
}

///**********************************************************

// Intersects each set of outputSets with each set of partitionSets and append to outputSets
void CUDAIntersectSiblingSets(
  PkPooledRawBitSetArray& outputSets
, const PkPooledRawBitSetArray& partitionSets
)
{
	PkLogf( "Begin CUDA intersection.\n" );

	// Assert buffers have matching bit set specifications
	PkAssert( outputSets.num_bits() == partitionSets.num_bits() );
	PkAssert( outputSets.num_blocks() == partitionSets.num_blocks() );

	// Begin timer
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);				

	// Early out if we don't have any sets to intersect
	if ( outputSets.size() <= 0 || partitionSets.size() <= 0 )
	{
		outputSets.clear();
		return;
	}

	// Store original locus sizes and block counts
	const size_t num_sib_sets_at_locus_a = outputSets.size();
	const size_t num_sib_sets_at_locus_b = partitionSets.size();
	const size_t num_bit_blocks_per_sib_set = outputSets.num_blocks();

	// Push first buffer to GPU
	PkCUDABlockType* gpu_sib_sets_matrix_a = CUDAPushSiblingSetsToGPU( outputSets );
	PkAssert( NULL != gpu_sib_sets_matrix_a );

	// Swizzle bits
	PkCUDABlockType* gpu_sib_sets_swizzled_matrix_a = CUDASwizzleSiblingSets(
		  gpu_sib_sets_matrix_a
		, num_sib_sets_at_locus_a
		, num_bit_blocks_per_sib_set
		);
	PkAssert( NULL != gpu_sib_sets_swizzled_matrix_a );

	// Push second buffer to GPU
	PkCUDABlockType* gpu_sib_sets_matrix_b = CUDAPushSiblingSetsToGPU( partitionSets );
	PkAssert( NULL != gpu_sib_sets_matrix_b );

	// Swizzle bits
	PkCUDABlockType* gpu_sib_sets_swizzled_matrix_b = CUDASwizzleSiblingSets(
		  gpu_sib_sets_matrix_b
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		);
	PkAssert( NULL != gpu_sib_sets_swizzled_matrix_b );

	// Prepare GPU intersection threads
	const unsigned int num_threads_in_block_x = Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_X;
	const unsigned int num_threads_in_block_y = Pk_CUDA_NUM_INTERSECTION_THREADS_IN_BLOCK_Y;
	const unsigned int num_blocks_x = num_sib_sets_at_locus_b / num_threads_in_block_x + ( ( num_sib_sets_at_locus_b % num_threads_in_block_x ) > 0 );
	const unsigned int num_blocks_y = num_sib_sets_at_locus_a / num_threads_in_block_y + ( ( num_sib_sets_at_locus_a % num_threads_in_block_y ) > 0 );
	const dim3 dimComputingGrid( num_blocks_x, num_blocks_y );
	const dim3 dimComputingBlock( num_threads_in_block_x, num_threads_in_block_y );

	// Free original (non-swizzled) sibling set matrices (these are blocking operations on the GPU)
	hipFree( gpu_sib_sets_matrix_b );
	hipFree( gpu_sib_sets_matrix_a );

	// Allocate GPU memory for intersection
	PkCUDABlockType* gpu_sib_sets_matrix_intersection = NULL;
	const size_t num_matrix_intersection_bytes = num_sib_sets_at_locus_a * num_sib_sets_at_locus_b * num_bit_blocks_per_sib_set * sizeof( PkCUDABlockType );
	hipMalloc((void**)&gpu_sib_sets_matrix_intersection, num_matrix_intersection_bytes );		
	PkAssert( NULL != gpu_sib_sets_matrix_intersection );

	// Execute GPU intersection kernel
	CUDASwizzledSiblingIntersectionKernel<<<dimComputingGrid,dimComputingBlock>>>(
		  gpu_sib_sets_swizzled_matrix_a
		, gpu_sib_sets_swizzled_matrix_b
		, gpu_sib_sets_matrix_intersection
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		);

	// Deallocate swizzled buffers
	hipDeviceSynchronize(); // @todo: not sure if this is necessary as hipFree may be blocking
	hipFree( gpu_sib_sets_swizzled_matrix_b );
	hipFree( gpu_sib_sets_swizzled_matrix_a );

	// Copy intersections back to CPU
	CUDAUnswizzleIntersectionMatrix(
		  outputSets
		, partitionSets
		, num_matrix_intersection_bytes
		, gpu_sib_sets_matrix_intersection
		, dimComputingGrid
		, dimComputingBlock
		);

// BEGIN INTEGRATION
	// Count bits for each bit set
	const PkCUDABitCountType * gpu_sib_sets_matrix_intersection_bit_counts = CUDACountIntersectionBits(
		  gpu_sib_sets_matrix_intersection
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		, dimComputingGrid
		, dimComputingBlock
		);
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_bit_counts );
	
	// Compute hash for each bit set
	const std::size_t * gpu_sib_sets_matrix_intersection_hashes = CUDAComputeIntersectionHashes(
		  gpu_sib_sets_matrix_intersection
		, gpu_sib_sets_matrix_intersection_bit_counts
		, num_sib_sets_at_locus_a
		, num_sib_sets_at_locus_b
		, num_bit_blocks_per_sib_set
		, dimComputingGrid
		, dimComputingBlock
		);
	PkAssert( NULL != gpu_sib_sets_matrix_intersection_hashes );
	
	// Copy hashes back to CPU, release from GPU
	const size_t num_data_values = num_sib_sets_at_locus_a * num_sib_sets_at_locus_b;

	const std::size_t * cpu_bit_hashes = CUDACopyFromGPUToCPUAndReleaseGPU<std::size_t>( gpu_sib_sets_matrix_intersection_hashes, num_data_values );
	PkAssert( NULL != cpu_bit_hashes );

	// Copy bit counts back to CPU, release from GPU
	const PkCUDABitCountType * cpu_bit_counts = CUDACopyFromGPUToCPUAndReleaseGPU<PkCUDABitCountType>( gpu_sib_sets_matrix_intersection_bit_counts, num_data_values );
	PkAssert( NULL != cpu_bit_counts );

	// Deallocate CPU data
	free( (void*)cpu_bit_hashes ); cpu_bit_hashes = NULL;
	free( (void*)cpu_bit_counts ); cpu_bit_counts = NULL;
// END INTEGRATION

	// Deallocate GPU intersection (blocks GPU)
	hipFree( gpu_sib_sets_matrix_intersection );

	// Stop timer
	cutStopTimer(timer);
	PkLogf( "End CUDA intersection (%d ms).\n", (unsigned int)cutGetTimerValue(timer) );
	cutDeleteTimer(timer);
}

} // end of Pk namespace

// BEGIN ORIGINAL SAMPLE CODE:

#if 0

#define SIBGROUP_SIZE 1024

// GPU kernel for calculating the intersection
__global__ void SimpleSiblingIntersection(unsigned int * gpu_sib_group_a,unsigned int * gpu_sib_group_b,unsigned int * gpu_sib_group_intersection){

	//Get position in the overall computation matrix
	unsigned int row=blockIdx.y*16+threadIdx.y;
	unsigned int column=blockIdx.x*16+threadIdx.x;

	//Compute the intersection
	for(int k=0; k<4; ++k)
		gpu_sib_group_intersection[SIBGROUP_SIZE*SIBGROUP_SIZE*k+SIBGROUP_SIZE*row+column]=gpu_sib_group_a[SIBGROUP_SIZE*k+row] & gpu_sib_group_b[SIBGROUP_SIZE*k+column];
}

// GPU kernel with shared memory for calculating the intersection
__global__ void SiblingIntersection(unsigned int * gpu_sib_group_a,unsigned int * gpu_sib_group_b,unsigned int * gpu_sib_group_intersection){
	
	//Shared memory used as cache
	__shared__ unsigned int sib_group_shared_a[16][4];	
	__shared__ unsigned int sib_group_shared_b[16][4];	

	//First row loads the sibgroup in shared cache 
	if (threadIdx.y==0){
		for (int k=0; k<4; ++k)
			sib_group_shared_a[threadIdx.x][k]=gpu_sib_group_a[SIBGROUP_SIZE*k+blockIdx.y*16+threadIdx.x];
		for (int k=0; k<4; ++k)
			sib_group_shared_b[threadIdx.x][k]=gpu_sib_group_b[SIBGROUP_SIZE*k+blockIdx.x*16+threadIdx.x];
	}
	__syncthreads();
	
	//Get position in the overall computation matrix
	unsigned int row=blockIdx.y*16+threadIdx.y;
	unsigned int column=blockIdx.x*16+threadIdx.x;

	//Compute the intersection on sibgroup in shared memory
	for(int k=0; k<4; ++k)
		gpu_sib_group_intersection[SIBGROUP_SIZE*SIBGROUP_SIZE*k+SIBGROUP_SIZE*row+column]
			=sib_group_shared_a[threadIdx.y][k] & sib_group_shared_b[threadIdx.x][k];
}

int CudaTestMain (int argc, char **argv) {
	
	//Allocate memory for locus_a and locus_b n=m=128 sibgroup=128bits
	unsigned int sibgroup_size=SIBGROUP_SIZE*4*sizeof(unsigned int);
	unsigned int * sib_group_a = (unsigned int *) malloc(sibgroup_size);
	unsigned int * sib_group_b = (unsigned int *) malloc(sibgroup_size);
	
	//Create random sibgroups
	unsigned int iseed = (unsigned int)time(NULL);
	srand (iseed);
	for (int i=0; i<SIBGROUP_SIZE*4; ++i){
		sib_group_a[i]=rand();
		sib_group_b[i]=rand();		
	}
	
	//Allocate memory for the intersection n*m=128*128 sibgroup=128bits
	unsigned int sibgroup_size_intersection=SIBGROUP_SIZE*SIBGROUP_SIZE*4*sizeof(unsigned int);
	unsigned int * sib_group_intersection = (unsigned int *) malloc(sibgroup_size_intersection);
	unsigned int * sib_group_intersection_copy = (unsigned int *) malloc(sibgroup_size_intersection);
	
	//Compute the intersection as 32bit binary operation
	unsigned int baseline_timer;
	cutCreateTimer(&baseline_timer);
	cutStartTimer(baseline_timer);			
	for (int i=0; i<SIBGROUP_SIZE; ++i)
		for (int j=0; j<SIBGROUP_SIZE; ++j)
			for (int k=0; k<4; ++k) 
				sib_group_intersection[SIBGROUP_SIZE*SIBGROUP_SIZE*k+SIBGROUP_SIZE*i+j]=sib_group_a[SIBGROUP_SIZE*k+i] & sib_group_b[SIBGROUP_SIZE*k+j];
	cutStopTimer(baseline_timer);	
	printf("CPU intersection took %d ms\n", (unsigned int)cutGetTimerValue(baseline_timer));	
	
	//Allocate GPU memory for locus_a and locus_b
	unsigned int * gpu_sib_group_a;
	unsigned int * gpu_sib_group_b;
	hipMalloc((void**)&gpu_sib_group_a,sibgroup_size);
	hipMalloc((void**)&gpu_sib_group_b,sibgroup_size);
	
	//Allocate GPU memory for intersection
	unsigned int * gpu_sib_group_intersection;	
	hipMalloc((void**)&gpu_sib_group_intersection,sibgroup_size_intersection);		
	
	//Copy sibling groups to memory
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);				
	hipMemcpy(gpu_sib_group_a,sib_group_a,sibgroup_size,hipMemcpyHostToDevice);	
	hipMemcpy(gpu_sib_group_b,sib_group_b,sibgroup_size,hipMemcpyHostToDevice);	

	//Prepare GPU threads (16x16) and block (8x8)
	dim3 dimComputingGrid(SIBGROUP_SIZE/16,SIBGROUP_SIZE/16);
	dim3 dimComputingBlock(16,16);		
		
	//Execute GPU kernel
	SimpleSiblingIntersection<<<dimComputingGrid,dimComputingBlock>>>(gpu_sib_group_a,gpu_sib_group_b,gpu_sib_group_intersection);
	
	//Copy back the intersection
	hipMemcpy(sib_group_intersection_copy,gpu_sib_group_intersection,sibgroup_size_intersection,hipMemcpyDeviceToHost);			
	cutStopTimer(timer);
	printf("GPU intersection took  %d ms\n",(unsigned int)cutGetTimerValue(timer));
	printf("Speed-up %.2fx\n",cutGetTimerValue(baseline_timer)/cutGetTimerValue(timer));	
	cutDeleteTimer(timer);
	
	//Execute GPU kernel
	cutCreateTimer(&timer);
	cutStartTimer(timer);	
	hipMemcpy(gpu_sib_group_a,sib_group_a,sibgroup_size,hipMemcpyHostToDevice);	
	hipMemcpy(gpu_sib_group_b,sib_group_b,sibgroup_size,hipMemcpyHostToDevice);	
	SiblingIntersection<<<dimComputingGrid,dimComputingBlock>>>(gpu_sib_group_a,gpu_sib_group_b,gpu_sib_group_intersection);
	
	//Copy back the intersection
	hipMemcpy(sib_group_intersection_copy,gpu_sib_group_intersection,sibgroup_size_intersection,hipMemcpyDeviceToHost);			
	cutStopTimer(timer);
	printf("GPU intersection took  %d ms\n",(unsigned int)cutGetTimerValue(timer));
	printf("Speed-up %.2fx\n",cutGetTimerValue(baseline_timer)/cutGetTimerValue(timer));	
	
	//Compare gpu and cpu result for correctness
	for (int i=0; i<SIBGROUP_SIZE*SIBGROUP_SIZE*4; ++i)
		if (sib_group_intersection[i]!=sib_group_intersection_copy[i]) {
			printf("Gpu and cpu produce different results.\n");
			printf("%d != %d (%d)\n",sib_group_intersection[i],sib_group_intersection_copy[i],i);
			break;
		}
	
	//Deallocate GPU data structures
	hipFree(gpu_sib_group_a);
	hipFree(gpu_sib_group_b);
	hipFree(gpu_sib_group_intersection);
	
	//Deallocate data structures
	free(sib_group_a);
	free(sib_group_b);
		
    return 0;
}

#endif // 0

// END ORIGINAL SAMPLE CODE
